#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <assert.h>
#include ".\cuda_include\helper_string.h"  // helper for shared functions common to CUDA Samples


#include <hipblas.h>
#include <hipsolver.h>
// CUDA and CUBLAS functions
//#include ".\cuda_include\helper_functions.h"
#include ".\cuda_include\hip/hip_runtime_api.h"

#include <stdio.h>




using Eigen::MatrixXd;
using Eigen::VectorXd;

using namespace Eigen;


__global__ void addKernel(int *c, const int *a, const int *b)
{
}

VectorXd PushAndQueue(double push, VectorXd A, double queue)
{
	VectorXd result(A.rows() + 2);
	result[0] = push;
	for (int i = 0; i < A.rows(); i++)
	{
		result[i] = A[i];
	}
	result[A.rows() + 1] = queue;
	return result;
}


int MethodOfLines::MoLiteration(double Tend, double Tdone, double dt, double *G, int GRows, int GCols, double *lamb, int lambRows, int lambCols, double inx2, double r, double K, MatrixXd A1, MatrixXd Aend, MatrixXd H)
{
	int count = 0;
	while (Tend - Tdone > 1E-8)
	{
		Tdone += dt;
		
		int sizeG = GRows * GCols;
		int sizeLamb = lambRows * lambCols;
		int memG = sizeof(double) * sizeG;
		int memLamb = sizeof(double) * sizeLamb;

		double *d_G, *d_lamb, *d_FFF;
		int sizeFFF = GRows * lambCols;
		int memFFF = sizeof(double)* sizeFFF;

		double *h_FFF = (double *)malloc(memFFF);
		double *h_CUBLAS = (double *)malloc(memFFF);

		checkCudaErrors(hipMalloc((void **)&d_G, memG));
		checkCudaErrors(hipMalloc((void **)&d_lamb, memLamb));
		checkCudaErrors(hipMemcpy(d_G, G, memG, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_lamb, lamb, memLamb, hipMemcpyHostToDevice));
		checkCudaErrors(hipMalloc((void **)&d_FFF, memFFF));

		hipblasHandle_t handle;
		checkCudaErrors(hipblasCreate(&handle));
		const double alpha = 1.0;
		const double beta = 1.0;
		checkCudaErrors(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, GRows, lambCols, GCols, &alpha, d_G, GRows, d_lamb, lambRows, &beta, d_FFF, GRows));
		
		checkCudaErrors(hipMemcpy(h_FFF, d_FFF, memFFF, hipMemcpyDeviceToHost));
		printf("after hipblasDgemm:\r\n");
		//double i[] = h_FFF;
		VectorXd FFF = Map<VectorXd >(h_FFF, GRows, lambCols);
		VectorXd fff = PushAndQueue(0, FFF, inx2 - exp(-r*Tdone)*K);
		printf("after PushAndQueue:\r\n");
		MatrixXd HH(A1.cols(), A1.cols());
		HH.row(0) = A1;
		HH.middleRows(1, HH.rows() - 2) = H;
		HH.row(HH.rows() - 1) = Aend;
		printf("after HH construction:\r\n");
		//LLT<MatrixXd> lltOfA(HH);
		//lamb = lltOfA.solve(fff);

		hipsolverHandle_t cusolverH = NULL; 
		hipblasHandle_t cublasH = NULL; 
		hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS; 
		hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS; 
		hipError_t cudaStat1 = hipSuccess; 
		hipError_t cudaStat2 = hipSuccess; 
		hipError_t cudaStat3 = hipSuccess; 
		hipError_t cudaStat4 = hipSuccess; 
		const int m = HH.rows(); const int lda = m; const int ldb = m; const int nrhs = 1; // number of right hand side vectors
		double *XC = new double[ldb*nrhs];
		
		double *d_A = NULL; // linear memory of GPU 
		double *d_tau = NULL; // linear memory of GPU 
		double *d_B = NULL; int *devInfo = NULL; // info in gpu (device copy) 
		double *d_work = NULL; 
		int lwork = 0; 
		int info_gpu = 0; 
		const double one = 1;

		cusolver_status = hipsolverDnCreate(&cusolverH); 
		assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status); 
		printf("after cusolver create:\r\n");
		cublas_status = hipblasCreate(&cublasH); 
		assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
		printf("after cublas create:\r\n");

		cudaStat1 = hipMalloc((void**)&d_A, sizeof(double) * lda * m); 
		cudaStat2 = hipMalloc((void**)&d_tau, sizeof(double) * m); 
		cudaStat3 = hipMalloc((void**)&d_B, sizeof(double) * ldb * nrhs); 
		cudaStat4 = hipMalloc((void**)&devInfo, sizeof(int)); 
		assert(hipSuccess == cudaStat1); 
		assert(hipSuccess == cudaStat2); 
		assert(hipSuccess == cudaStat3); 
		assert(hipSuccess == cudaStat4); 
		cudaStat1 = hipMemcpy(d_A, HH.data(), sizeof(double) * lda * m, hipMemcpyHostToDevice); 
		cudaStat2 = hipMemcpy(d_B, fff.data(), sizeof(double) * ldb * nrhs, hipMemcpyHostToDevice); 
		assert(hipSuccess == cudaStat1); assert(hipSuccess == cudaStat2);

		// step 3: query working space of geqrf and ormqr 
		cusolver_status = hipsolverDnDgeqrf_bufferSize( cusolverH, m, m, d_A, lda, &lwork); 
		assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS); 
		cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork); 
		printf("after initialisation:\r\n");
		assert(hipSuccess == cudaStat1); 
		// step 4: compute QR factorization 
		cusolver_status = hipsolverDnDgeqrf( cusolverH, m, m, d_A, lda, d_tau, d_work, lwork, devInfo); 
		cudaStat1 = hipDeviceSynchronize(); 
		assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status); 
		assert(hipSuccess == cudaStat1); 
		printf("after QR factorization:\r\n");
		// check if QR is good or not 
		cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost); 
		assert(hipSuccess == cudaStat1); 
		printf("after geqrf: info_gpu = %d\n", info_gpu); 
		assert(0 == info_gpu); 
		// step 5: compute Q^T*B 
		cusolver_status= hipsolverDnDormqr( cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, m, nrhs, m, d_A, lda, d_tau, d_B, ldb, d_work, lwork, devInfo); 
		cudaStat1 = hipDeviceSynchronize(); 
		assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status); 
		assert(hipSuccess == cudaStat1);

		// check if QR is good or not 
		cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost); 
		assert(hipSuccess == cudaStat1); 
		printf("after ormqr: info_gpu = %d\n", info_gpu); 
		assert(0 == info_gpu); 
		// step 6: compute x = R \ Q^T*B 
		cublas_status = hipblasDtrsm( cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, nrhs, &one, d_A, lda, d_B, ldb); 
		cudaStat1 = hipDeviceSynchronize(); assert(HIPBLAS_STATUS_SUCCESS == cublas_status); 
		assert(hipSuccess == cudaStat1); 
		cudaStat1 = hipMemcpy(XC, d_B, sizeof(double)*ldb*nrhs, hipMemcpyDeviceToHost); 
		assert(hipSuccess == cudaStat1); 
		
		/*printf("X = (matlab base-1)\n"); 
		printMatrix(m, nrhs, XC, ldb, "X"); */

		// free resources 
		if (d_A ) hipFree(d_A); 
		if (d_tau ) hipFree(d_tau); 
		if (d_B ) hipFree(d_B); 
		if (devInfo) hipFree(devInfo); 
		if (d_work ) hipFree(d_work); 
		if (cublasH ) hipblasDestroy(cublasH); 
		if (cusolverH) hipsolverDnDestroy(cusolverH); 
		hipDeviceReset();

		
		
		count++;
		printf("%i\r\n", count);
	}
    return 0;
}

void main()
{}

