#include "Gaussian2d2.h"

namespace Leicester
{
	namespace ThrustLib
	{
		Gaussian2d2::Gaussian2d2()
		{
		}

		Gaussian2d2::~Gaussian2d2()
		{
			//this->testNodes.clear();
			//this->testNodes.shrink_to_fit();
			//this->centralNodes.clear();
			//this->centralNodes.shrink_to_fit();
		}

		Gaussian2d2::Gaussian2d2(double tLower, double tUpper, const double* N)
		{
			hipStream_t s1, s2;
			hipStreamCreate(&s1);
			hipStreamCreate(&s2);

			double length = N[0] * N[1];
			this->rows = length;
			this->cols = length;
			
			int sizeTestNodes = (2 + 2 * length) * sizeof(double);
			double * h_TXYZ = (double*)malloc(sizeTestNodes);
			double * d_TXYZ;
			hipMalloc((void **)&d_TXYZ, sizeTestNodes);
			//pinnedVector ph_testNodes(h_TXYZ, h_TXYZ + sizeTestNodes);
			//device_ptr<double> dv_tp;
			//double* d_tp = thrust::raw_pointer_cast(dv_tp);
			//double* h_tp = thrust::raw_pointer_cast(ph_testNodes.data());
			hipMemcpyAsync(d_TXYZ, h_TXYZ, sizeTestNodes, hipMemcpyHostToDevice, s1);
			hipStreamSynchronize(s1);
			hipError_t e = hipPeekAtLastError();

			int sizeN = 4 * sizeof(double);
			pinnedVector ph_N(N, N + sizeN);
			device_vector<double> dv_N(sizeN);
			hipMemcpyAsync(thrust::raw_pointer_cast(dv_N.data()), thrust::raw_pointer_cast(ph_N.data()),
				dv_N.size() * sizeof(double), hipMemcpyHostToDevice, s2);


			hipStreamSynchronize(s2);
			e = hipPeekAtLastError();
			hipDeviceSynchronize();
			
			//double* d_TP = thrust::raw_pointer_cast(dv_tp);
			
			GenerateTestNodes << <1, 1 >> > (tLower, tUpper, thrust::raw_pointer_cast(dv_N.data()), d_TXYZ);

			device_ptr<double> dv_tp = thrust::device_pointer_cast<double>(d_TXYZ);
			double * d_CN;
			hipMalloc((void **)&d_CN, sizeTestNodes);
			
			//thrust::copy(thrust::cuda::par.on(s1), dv_tp, dv_tp + 2 + (2*length), dv_cn);
			hipMemcpyAsync(d_CN, d_TXYZ, sizeTestNodes, hipMemcpyHostToDevice, s1);

			hipStreamSynchronize(s1);
			e = hipPeekAtLastError();
			hipDeviceSynchronize();
			
			device_ptr<double> dv_cn(d_CN);
			
			this->testNodes = dv_tp;
			this->centralNodes = dv_cn;
		}


		typedef thrust::host_vector<double, thrust::cuda::experimental::pinned_allocator<double>> pinnedVector;

		vector<MatrixXd> Gaussian2d2::Gaussian2d(const MatrixXd & CN, const MatrixXd & A, const MatrixXd & C)
		{
			//hipStream_t s1, s2;
			//hipStreamCreate(&s1);
			//hipStreamCreate(&s2);
			//
			//double length = N[3] * N[2];
			//this->cols = length;

			//int sizeCentralNodes = 2 * length * sizeof(double);
			//double * h_TXYZ = (double*)malloc(sizeCentralNodes);
			//pinnedVector ph_centralNodes(h_TXYZ, h_TXYZ + sizeCentralNodes);
			//device_vector<double> dv_cn(sizeCentralNodes);
			//hipMemcpyAsync(thrust::raw_pointer_cast(dv_cn.data()), thrust::raw_pointer_cast(ph_centralNodes.data()),
			//	dv_cn.size() * sizeof(double), hipMemcpyHostToDevice, s1);

			//int sizeN = 4 * sizeof(double);
			//pinnedVector ph_N(N, N + sizeN);
			//device_vector<double> dv_N(sizeN);
			//hipMemcpyAsync(thrust::raw_pointer_cast(dv_N.data()), thrust::raw_pointer_cast(ph_N.data()),
			//	dv_N.size() * sizeof(double), hipMemcpyHostToDevice, s2);

			//hipStreamSynchronize(s1);
			//hipStreamSynchronize(s2);
			////hipDeviceSynchronize();

			//double* d_CN = thrust::raw_pointer_cast(dv_cn.data());

			//GenerateTestNodes << <1, 1 >> > (tLower, tUpper, thrust::raw_pointer_cast(dv_N.data()), d_CN);

			//printf("CN:\r\n");
			//host_vector<double> h_cn(dv_cn.size());
			//thrust::copy(dv_cn.begin(), dv_cn.end(), h_cn.begin());
			//double *raw_cn = thrust::raw_pointer_cast(h_cn.data());
			//Utility::printMatrix(raw_cn, dim3(2, 15));

			hipStream_t s1;
			hipStreamCreate(&s1);
			this->cols = CN.rows();

			int sizeCentralNodes = sizeof(double) * CN.rows() * CN.cols();
			const double *h_centralNodes = CN.data();
			/*pinnedVector ph_centralNodes(h_centralNodes, h_centralNodes + sizeCentralNodes);
			device_vector<mytype> pd_centralNodes(sizeCentralNodes);
			hipMemcpyAsync(thrust::raw_pointer_cast(pd_centralNodes.data()), thrust::raw_pointer_cast(ph_centralNodes.data()),
			pd_centralNodes.size() * sizeof(mytype), hipMemcpyHostToDevice, s1)
			this->centralNodes = pd_centralNodes;*/

			double *d_centralNodes;
			hipMalloc((void**)&d_centralNodes, sizeCentralNodes);
			device_ptr<double> dp_centralNodes = thrust::device_pointer_cast<double>(d_centralNodes);
			hipMemcpyAsync(d_centralNodes, h_centralNodes, sizeCentralNodes, hipMemcpyKind::hipMemcpyHostToDevice, s1);

			hipDeviceSynchronize();

			//device_vector<double> dv_centralNodes(dp_centralNodes, dp_centralNodes + (CN.rows() * CN.cols()));
			this->centralNodes = dp_centralNodes;

			return Gaussian2d(A, C);

		}

		vector<MatrixXd> Gaussian2d2::Gaussian2d(const MatrixXd & A, const MatrixXd & C)
		{
			hipStream_t s1, s2, s3, s4;
			hipStreamCreate(&s1);
			hipStreamCreate(&s2);
			hipStreamCreate(&s3);
			hipStreamCreate(&s4);

			const double *h_a = A.data();
			const double *h_c = C.data();
			device_vector<double> d_tp(this->testNodes, this->testNodes + 2 + (rows * 2));
			device_vector<double> d_cn(this->centralNodes, this->centralNodes + 2 + (cols * 2));

			device_vector<double> d_tp0(rows);
			device_vector<double> d_cn0(cols);
			thrust::copy(thrust::cuda::par.on(s1), d_tp.begin() + 2, d_tp.begin() + 2 + rows, d_tp0.begin()); //first column
			thrust::copy(thrust::cuda::par.on(s1), d_cn.begin() + 2, d_cn.begin() + 2 + cols, d_cn0.begin()); //first column

			device_vector<double> d_tp1(rows);
			device_vector<double> d_cn1(rows);
			thrust::copy(thrust::cuda::par.on(s2), d_tp.begin() + 2 + rows, d_tp.begin() + 2 + (2 * rows), d_tp1.begin()); //second column
			thrust::copy(thrust::cuda::par.on(s2), d_cn.begin() + 2 + cols, d_cn.begin() + 2 + (2 * cols), d_cn1.begin()); //second column

			device_vector<double> d_a(h_a, h_a + (A.rows() * A.cols()));
			device_vector<double> d_c(h_c, h_c + (C.rows() * C.cols()));

			device_vector<double> d_PHI1(rows * cols);

			hipStreamSynchronize(s1);
			hipStreamSynchronize(s2);
			counting_iterator<int> first(0);
			counting_iterator<int> last(rows * cols);


			//thrust::transform(thrust::cuda::par.on(s1), d_PHI1.begin(), d_PHI1.end(), d_PHI1.begin(), 
			//	phi1_functor2(raw_pointer_cast(d_tp0.data()), A(0, 0), raw_pointer_cast(d_cn0.data()), C(0, 0), rows, cols));
			thrust::for_each(thrust::cuda::par.on(s1),
				thrust::make_zip_iterator(
					thrust::make_tuple(first, d_PHI1.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(last, d_PHI1.end())),
				phi_functor3(raw_pointer_cast(d_tp0.data()), A(0, 0), raw_pointer_cast(d_cn0.data()), C(0, 0), rows, cols)
			);
			device_vector<double> d_PHI2(rows * cols);

			//thrust::transform(thrust::cuda::par.on(s2), d_PHI2.begin(), d_PHI2.end(), d_PHI2.begin(), 
			//	phi2_functor2(raw_pointer_cast(d_tp1.data()), A(0, 1), raw_pointer_cast(d_cn1.data()), C(0, 1), rows, cols));
			thrust::for_each(thrust::cuda::par.on(s2),
				thrust::make_zip_iterator(
					thrust::make_tuple(first, d_PHI2.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(last, d_PHI2.end())),
				phi_functor3(raw_pointer_cast(d_tp1.data()), A(0, 1), raw_pointer_cast(d_cn1.data()), C(0, 1), rows, cols)
			);

			device_vector<double> d_D(rows * cols);
			hipStreamSynchronize(s1);
			hipStreamSynchronize(s2);
			thrust::transform(thrust::cuda::par.on(s1), d_PHI1.begin(), d_PHI1.end(), d_PHI2.begin(), d_D.begin(),
				d_functor2());
			hipStreamSynchronize(s1);

			d_PHI1.clear();
			d_PHI1.shrink_to_fit();
			d_PHI2.clear();
			d_PHI2.shrink_to_fit();

			//Calculate Dt
			device_vector<double> d_Dt(rows * cols);
			double scalarDt = -2 * (A(0, 0) / C(0, 0)) * (A(0, 0) / C(0, 0));
			//thrust::transform(thrust::cuda::par.on(s1), d_Dt.begin(), d_Dt.end(), d_D.begin(), d_Dt.begin(), 
			//	dt_functor2(raw_pointer_cast(d_tp0.data()), scalarDt, raw_pointer_cast(d_cn0.data()), rows, cols));
			thrust::for_each(thrust::cuda::par.on(s1),
				thrust::make_zip_iterator(
					thrust::make_tuple(first, d_D.begin(), d_Dt.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(last, d_D.end(), d_Dt.end())),
				dt_functor3(raw_pointer_cast(d_tp0.data()), scalarDt, raw_pointer_cast(d_cn0.data()), rows, cols)
			);

			//Calculate Dx
			device_vector<double> d_Dx(rows * cols);
			double scalarDx = -2 * ((A(0, 1) / C(0, 1)) * (A(0, 1) / C(0, 1)));
			/*thrust::transform(thrust::cuda::par.on(s2), d_Dx.begin(), d_Dx.end(), d_D.begin(), d_Dx.begin(),
			dx_functor2(raw_pointer_cast(d_tp1.data()), scalarDx, raw_pointer_cast(d_cn1.data()), rows, cols));*/
			thrust::for_each(thrust::cuda::par.on(s2),
				thrust::make_zip_iterator(
					thrust::make_tuple(first, d_D.begin(), d_Dx.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(last, d_D.end(), d_Dx.end())),
				dx_functor3(raw_pointer_cast(d_tp1.data()), scalarDx, raw_pointer_cast(d_cn1.data()), rows, cols)
			);
			//Calculate Dxx
			device_vector<double> d_Dxx(rows * cols);
			double sA = A(0, 1) * A(0, 1);
			double qA = sA * sA;
			double sC = C(0, 1) * C(0, 1);
			double qC = sC * sC;
			double scalarDxx1 = 4 * qA / qC;
			double scalarDxx2 = -2 * sA / sC;
			/*thrust::transform(thrust::cuda::par.on(s3), d_Dxx.begin(), d_Dxx.end(), d_D.begin(), d_Dxx.begin(),
			dxx_functor2(raw_pointer_cast(d_tp1.data()), scalarDxx1, scalarDxx2, raw_pointer_cast(d_cn1.data()), rows, cols));*/
			thrust::for_each(thrust::cuda::par.on(s3),
				thrust::make_zip_iterator(
					thrust::make_tuple(first, d_D.begin(), d_Dxx.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(last, d_D.end(), d_Dxx.end())),
				dxx_functor3(raw_pointer_cast(d_tp1.data()), scalarDxx1, scalarDxx2, raw_pointer_cast(d_cn1.data()), rows, cols)
			);
			hipStreamSynchronize(s1);
			hipStreamSynchronize(s2);
			hipStreamSynchronize(s3);
			hipStreamSynchronize(s4);


			//hipDeviceSynchronize();
			//double *h_Phi1 = (double*)malloc(sizeof(double) * rows * cols);
			//double *p_Phi1 = d_PHI1.data().get();
			//hipError_t e = hipMemcpy(h_Phi1, p_Phi1, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			//if (e != hipSuccess)
			//	printf("hipMemcpy h_Phi1 returned error %s (code %d), line(%d) when copying%i\n",
			//		hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);
			//Eigen::Map<Eigen::MatrixXd> dataMapPhi1(h_Phi1, rows, cols);
			//MatrixXd Phi1 = dataMapPhi1.eval();

			//hipDeviceSynchronize();
			//double *h_Phi2 = (double*)malloc(sizeof(double) * rows * cols);
			//double *p_Phi2 = d_PHI2.data().get();
			//e = hipMemcpy(h_Phi2, p_Phi2, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			//if (e != hipSuccess)
			//	printf("hipMemcpy h_Phi2 returned error %s (code %d), line(%d) when copying%i\n",
			//		hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);
			//Eigen::Map<Eigen::MatrixXd> dataMapPhi2(h_Phi2, rows, cols);
			//MatrixXd Phi2 = dataMapPhi2.eval();

			hipDeviceSynchronize();
			double *h_D = (double*)malloc(sizeof(double) * rows * cols);
			double *p_D = d_D.data().get();
			//hipError_t e = hipMemcpy(h_D, p_D, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			hipError_t e = hipMemcpyAsync(h_D, p_D, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost, s1);
			if (e != hipSuccess)
				printf("hipMemcpy h_D returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);


			hipDeviceSynchronize();
			double *h_Dt = (double*)malloc(sizeof(double) * rows * cols);
			double *p_Dt = d_Dt.data().get();
			//e = hipMemcpy(h_Dt, p_Dt, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			e = hipMemcpyAsync(h_Dt, p_Dt, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost, s2);
			if (e != hipSuccess)
				printf("hipMemcpy h_Dt returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);


			//hipDeviceSynchronize();
			double *h_Dx = (double*)malloc(sizeof(double) * rows * cols);
			double *p_Dx = d_Dx.data().get();
			//e = hipMemcpy(h_Dx, p_Dx, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			e = hipMemcpyAsync(h_Dx, p_Dx, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost, s3);
			if (e != hipSuccess)
				printf("hipMemcpy h_Dx returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);


			//hipDeviceSynchronize();
			double *h_Dxx = (double*)malloc(sizeof(double) * rows * cols);
			double *p_Dxx = d_Dxx.data().get();
			//e = hipMemcpy(h_Dxx, p_Dxx, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			e = hipMemcpyAsync(h_Dxx, p_Dxx, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost, s4);
			if (e != hipSuccess)
				printf("hipMemcpy h_Dxx returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);


			hipStreamSynchronize(s1);
			hipStreamSynchronize(s2);
			hipStreamSynchronize(s3);
			hipStreamSynchronize(s4);
			Eigen::Map<Eigen::MatrixXd> dataMapDxx(h_Dxx, rows, cols);
			MatrixXd Dxx = dataMapDxx.eval();
			Eigen::Map<Eigen::MatrixXd> dataMapDx(h_Dx, rows, cols);
			MatrixXd Dx = dataMapDx.eval();
			Eigen::Map<Eigen::MatrixXd> dataMapDt(h_Dt, rows, cols);
			MatrixXd Dt = dataMapDt.eval();
			Eigen::Map<Eigen::MatrixXd> dataMapD(h_D, rows, cols);
			MatrixXd D = dataMapD.eval();

			return{ D, Dt, Dx, Dxx };

		}
	}
}