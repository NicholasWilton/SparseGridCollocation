#include "GaussianNd1.h"

using Eigen::MatrixXd;
using Eigen::VectorXd;
using namespace std;
using namespace Eigen;
using namespace thrust;

namespace Leicester
{
	namespace ThrustLib
	{
		GaussianNd1::GaussianNd1(MatrixXd testNodes, MatrixXd centralNodes)
		{
			this->rows = testNodes.rows();
			const double *h_testNodes = testNodes.data();
			device_vector<double> d_testNodes(h_testNodes, h_testNodes + (testNodes.rows() * testNodes.cols()));
			this->testNodes = d_testNodes;
			this->cols = centralNodes.rows();
			const double *h_centralNodes = centralNodes.data();
			device_vector<double> d_centralNodes(h_centralNodes, h_centralNodes + (centralNodes.rows() * centralNodes.cols()));
			this->centralNodes = d_centralNodes;
			this->dimensions = testNodes.cols();
		}

		GaussianNd1::GaussianNd1(MatrixXd testNodes)
		{
			this->rows = testNodes.rows();
			const double *h_testNodes = testNodes.data();
			device_vector<double> d_testNodes(h_testNodes, h_testNodes + (testNodes.rows() * testNodes.cols()));
			this->testNodes = d_testNodes;
			this->dimensions = testNodes.cols();
		}

		GaussianNd1::~GaussianNd1()
		{
			this->testNodes.clear();
			this->testNodes.shrink_to_fit();
			this->centralNodes.clear();
			this->centralNodes.shrink_to_fit();
		}

		vector<MatrixXd> GaussianNd1::GaussianNd(const MatrixXd & CN, const MatrixXd & A, const MatrixXd & C)
		{
			this->cols = CN.rows();
			const double *h_centralNodes = CN.data();
			device_vector<double> d_centralNodes(h_centralNodes, h_centralNodes + (CN.rows() * CN.cols()));
			this->centralNodes = d_centralNodes;
			return this->GaussianNd(A, C);
		}

		vector<MatrixXd> GaussianNd1::GaussianNd(const MatrixXd & A, const MatrixXd & C)
		{
			hipStream_t s1, s2, s3;
			hipStreamCreate(&s1);
			hipStreamCreate(&s2);
			hipStreamCreate(&s3);

			const double *h_a = A.data();
			const double *h_c = C.data();


			device_vector<double> d_a(h_a, h_a + (A.rows() * A.cols()));
			device_vector<double> d_c(h_c, h_c + (C.rows() * C.cols()));

			device_vector<double> d_D(rows * cols);
			device_vector<double> d_Dt(rows * cols);
			device_vector<double> d_Dx(rows * cols);
			device_vector<double> d_Dxx(rows * cols);


			hipStreamSynchronize(s1);
			hipStreamSynchronize(s2);
			counting_iterator<int> first(0);
			counting_iterator<int> last(rows * cols);

			
			thrust::for_each(thrust::cuda::par.on(s1),
				thrust::make_zip_iterator(
					thrust::make_tuple(first, d_D.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(last, d_D.end())),
				phi_functor3Nd(raw_pointer_cast(testNodes.data()), raw_pointer_cast(d_a.data()), 
					raw_pointer_cast(centralNodes.data()), raw_pointer_cast(d_c.data()), rows, cols, dimensions)
			);

		

			hipStreamSynchronize(s1);
			

			//Calculate Dt
			double scalarDt = -2 * (A(0, 0) / C(0, 0)) * (A(0, 0) / C(0, 0));
			thrust::for_each(thrust::cuda::par.on(s1),
				thrust::make_zip_iterator(
					thrust::make_tuple(first, d_D.begin(), d_Dt.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(last, d_D.end(), d_Dt.end())),
				dt_functor3Nd(raw_pointer_cast(testNodes.data()), scalarDt, raw_pointer_cast(centralNodes.data()), rows, cols)
			);

			//Calculate Dx
			thrust::for_each(thrust::cuda::par.on(s2),
				thrust::make_zip_iterator(
					thrust::make_tuple(first, d_D.begin(), d_Dx.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(last, d_D.end(), d_Dx.end())),
				dx_functor3Nd(raw_pointer_cast(testNodes.data()), raw_pointer_cast(d_a.data()), 
					raw_pointer_cast(centralNodes.data()), raw_pointer_cast(d_c.data()), rows, cols, dimensions)
			);
			//Calculate Dxx
			thrust::for_each(thrust::cuda::par.on(s3),
				thrust::make_zip_iterator(
					thrust::make_tuple(first, d_D.begin(), d_Dxx.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(last, d_D.end(), d_Dxx.end())),
				dxx_functor3Nd(raw_pointer_cast(testNodes.data()), raw_pointer_cast(d_a.data()), raw_pointer_cast(d_c.data()),
					raw_pointer_cast(centralNodes.data()), rows, cols, dimensions)
			);
			hipStreamSynchronize(s1);
			hipStreamSynchronize(s2);
			hipStreamSynchronize(s3);


			//hipDeviceSynchronize();
			//double *h_Phi = (double*)malloc(sizeof(double) * rows * cols);
			//double *p_Phi = d_PHI.data().get();
			//hipError_t e = hipMemcpy(h_Phi, p_Phi, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			//if (e != hipSuccess)
			//	printf("hipMemcpy h_Phi returned error %s (code %d), line(%d) when copying%i\n",
			//		hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);
			//Eigen::Map<Eigen::MatrixXd> dataMapPhi(h_Phi, rows, cols);
			//MatrixXd Phi = dataMapPhi.eval();

			//hipDeviceSynchronize();
			//double *h_Phi2 = (double*)malloc(sizeof(double) * rows * cols);
			//double *p_Phi2 = d_PHI2.data().get();
			//e = hipMemcpy(h_Phi2, p_Phi2, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			//if (e != hipSuccess)
			//	printf("hipMemcpy h_Phi2 returned error %s (code %d), line(%d) when copying%i\n",
			//		hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);
			//Eigen::Map<Eigen::MatrixXd> dataMapPhi2(h_Phi2, rows, cols);
			//MatrixXd Phi2 = dataMapPhi2.eval();

			hipDeviceSynchronize();
			double *h_D = (double*)malloc(sizeof(double) * rows * cols);
			double *p_D = d_D.data().get();
			hipError_t e = hipMemcpy(h_D, p_D, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			if (e != hipSuccess)
				printf("hipMemcpy h_D returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);
			Eigen::Map<Eigen::MatrixXd> dataMapD(h_D, rows, cols);
			MatrixXd D = dataMapD.eval();

			//hipDeviceSynchronize();
			double *h_Dt = (double*)malloc(sizeof(double) * rows * cols);
			double *p_Dt = d_Dt.data().get();
			e = hipMemcpy(h_Dt, p_Dt, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			if (e != hipSuccess)
				printf("hipMemcpy h_Dt returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);
			Eigen::Map<Eigen::MatrixXd> dataMapDt(h_Dt, rows, cols);
			MatrixXd Dt = dataMapDt.eval();

			//hipDeviceSynchronize();
			double *h_Dx = (double*)malloc(sizeof(double) * rows * cols);
			double *p_Dx = d_Dx.data().get();
			e = hipMemcpy(h_Dx, p_Dx, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			if (e != hipSuccess)
				printf("hipMemcpy h_Dx returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);
			Eigen::Map<Eigen::MatrixXd> dataMapDx(h_Dx, rows, cols);
			MatrixXd Dx = dataMapDx.eval();

			//hipDeviceSynchronize();
			double *h_Dxx = (double*)malloc(sizeof(double) * rows * cols);
			double *p_Dxx = d_Dxx.data().get();
			e = hipMemcpy(h_Dxx, p_Dxx, sizeof(double) * rows * cols, hipMemcpyKind::hipMemcpyDeviceToHost);
			if (e != hipSuccess)
				printf("hipMemcpy h_Dxx returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * cols);
			Eigen::Map<Eigen::MatrixXd> dataMapDxx(h_Dxx, rows, cols);
			MatrixXd Dxx = dataMapDxx.eval();
			
			d_a.clear();
			d_a.shrink_to_fit();
			d_c.clear();
			d_c.shrink_to_fit();

			d_D.clear();
			d_D.shrink_to_fit();
			d_Dt.clear();
			d_Dt.shrink_to_fit();
			d_Dx.clear();
			d_Dx.shrink_to_fit();
			d_Dxx.clear();
			d_Dxx.shrink_to_fit();

			return{ D, Dt, Dx, Dxx };
			

		}

		vector<MatrixXd> GaussianNd1::GaussianNd(const MatrixXd & TP, const MatrixXd & CN, const MatrixXd & A, const MatrixXd & C)
		{
			int rows = TP.rows();
			const double *h_tp0 = TP.col(0).data();
			const double *h_tp1 = TP.col(1).data();
			const double *h_cn = CN.data();
			const double *h_a = A.data();
			const double *h_c = C.data();
			//wcout << printMatrix(TP.col(0)) << endl;
			//wcout << printMatrix(TP.col(1)) << endl;

			device_vector<double> d_tp0(h_tp0, h_tp0 + (TP.rows()));
			device_vector<double> d_tp1(h_tp1, h_tp1 + (TP.rows()));
			device_vector<double> d_cn(h_cn, h_cn + (CN.rows() * CN.cols()));
			device_vector<double> d_a(h_a, h_a + (A.rows() * A.cols()));
			device_vector<double> d_c(h_c, h_c + (C.rows() * C.cols()));

			device_vector<double> d_PHI1(rows * rows);
			device_vector<double> d_PHI2(rows * rows);
			device_vector<double> d_D(rows * rows);
			device_vector<double> d_Dt(rows * rows);
			device_vector<double> d_Dx(rows * rows);
			device_vector<double> d_Dxx(rows * rows);

			device_vector<double> d_test1(rows * rows);
			device_vector<double> d_test2(rows * rows);

			//thrust::transform(d_PHI1.begin(), d_PHI1.end(), d_PHI1.begin(), phi_functor2(d_tp0.data(), A(0, 0), CN.col(0).data(), C(0, 0), rows, TP.cols()));

			for (int i = 0; i < rows; i++)
			{
				//if (i == 0)
				//{
				//	d_test1.insert(d_test1.begin(), d_tp0.begin(), d_tp0.end());
				//	d_test2.insert(d_test2.begin(), d_tp1.begin(), d_tp1.end());
				//}
				//Calculate Phi1 & Phi2
				thrust::device_vector<double> phi1(rows);
				thrust::transform(d_tp0.begin(), d_tp0.end(), phi1.begin(), phi_functor(A(0, 0), CN(i, 0), C(0, 0)));
				d_PHI1.insert(d_PHI1.begin(), phi1.begin(), phi1.end());

				thrust::device_vector<double> phi2(rows);
				thrust::transform(d_tp1.begin(), d_tp1.end(), phi2.begin(), phi_functor(A(0, 1), CN(i, 1), C(0, 1)));
				d_PHI2.insert(d_PHI2.begin(), phi2.begin(), phi2.end());

				//Calculate D
				thrust::device_vector<double> d(rows);
				thrust::transform(phi1.begin(), phi1.end(), phi2.begin(), d.begin(), thrust::multiplies<double>());
				d_D.insert(d_D.begin(), d.begin(), d.end());

				//Calculate Dt
				thrust::device_vector<double> a1(rows);
				double scalarDt = -2 * (A(0, 0) / C(0, 0)) * (A(0, 0) / C(0, 0));
				thrust::transform(d_tp0.begin(), d_tp0.end(), a1.begin(), scalarVectorDifference_functor(scalarDt, CN(i, 0)));

				thrust::device_vector<double> b1(rows);
				thrust::transform(phi1.begin(), phi1.end(), a1.begin(), b1.begin(), thrust::multiplies<double>());

				thrust::device_vector<double> dt(rows);
				thrust::transform(phi2.begin(), phi2.end(), b1.begin(), dt.begin(), thrust::multiplies<double>());
				d_Dt.insert(d_Dt.begin(), dt.begin(), dt.end());
				a1.clear();
				a1.shrink_to_fit();
				b1.clear();
				b1.shrink_to_fit();

				//Calculate Dx
				double scalarDx = -2 * ((A(0, 1) / C(0, 1)) * (A(0, 1) / C(0, 1)));
				thrust::device_vector<double> a2(rows);
				thrust::transform(d_tp1.begin(), d_tp1.end(), a2.begin(), scalarVectorDifference_functor(scalarDx, CN(i, 1)));

				thrust::device_vector<double> b2(rows);
				thrust::transform(phi1.begin(), phi1.end(), a2.begin(), b2.begin(), thrust::multiplies<double>());

				thrust::device_vector<double> c2(rows);
				thrust::transform(phi2.begin(), phi2.end(), b2.begin(), c2.begin(), thrust::multiplies<double>());

				thrust::device_vector<double> dx(rows);
				thrust::transform(d_tp1.begin(), d_tp1.end(), c2.begin(), dx.begin(), thrust::multiplies<double>());
				d_Dx.insert(d_Dx.begin(), dx.begin(), dx.end());
				a2.clear();
				a2.shrink_to_fit();
				b2.clear();
				b2.shrink_to_fit();
				c2.clear();
				c2.shrink_to_fit();
				dx.clear();
				dx.shrink_to_fit();
				//Calculate Dxx
				double sA = A(0, 1) * A(0, 1);
				double qA = sA * sA;
				double sC = C(0, 1) * C(0, 1);
				double qC = sC * sC;

				thrust::device_vector<double> dTpCn(rows);
				thrust::transform(d_tp1.begin(), d_tp1.end(), dTpCn.begin(), vectorScalarDifference_functor(CN(i, 1)));

				thrust::device_vector<double> sDTpCn(rows);
				thrust::transform(dTpCn.begin(), dTpCn.end(), dTpCn.begin(), sDTpCn.begin(), thrust::multiplies<double>());

				thrust::device_vector<double> a3(rows);
				double scalarDxx1 = 4 * qA / qC;

				thrust::transform(sDTpCn.begin(), sDTpCn.end(), a3.begin(), vectorScalarMultiply_functor(scalarDxx1));

				thrust::device_vector<double> b3(rows);
				double scalarDxx2 = -2 * sA / sC;

				thrust::transform(a3.begin(), a3.end(), b3.begin(), vectorAddScalar_functor(scalarDxx2));

				thrust::device_vector<double> c3(rows);
				thrust::transform(d.begin(), d.end(), b3.begin(), c3.begin(), thrust::multiplies<double>());

				thrust::device_vector<double> sTpCol1(rows);
				thrust::transform(d_tp1.begin(), d_tp1.end(), d_tp1.begin(), sTpCol1.begin(), thrust::multiplies<double>());

				thrust::device_vector<double> dxx(rows);
				//thrust::device_vector<double> rsTpCol(rows);
				//thrust::copy(sTpCol1.rbegin(), sTpCol1.rend(), rsTpCol.begin());
				thrust::transform(c3.begin(), c3.end(), sTpCol1.begin(), dxx.begin(), thrust::multiplies<double>());
				d_Dxx.insert(d_Dxx.begin(), dxx.begin(), dxx.end());

				//d_test1.insert(d_test1.begin(), d_tp1.begin(), d_tp1.end());
				dTpCn.clear();
				dTpCn.shrink_to_fit();
				sDTpCn.clear();
				sDTpCn.shrink_to_fit();
				b3.clear();
				b3.shrink_to_fit();
				c3.clear();
				c3.shrink_to_fit();
				sTpCol1.clear();
				sTpCol1.shrink_to_fit();

				/*if (i == 1)
				{
				d_test1.insert(d_test1.begin(), d_tp1.begin(), d_tp1.end());
				d_test2.insert(d_test2.begin(), d_Dxx.begin(), d_Dxx.end());
				}*/
			}

			//printf("Phi1:\r\n");
			//host_vector<double> h_PHI1(d_PHI1.size());
			//thrust::copy(d_PHI1.begin(), d_PHI1.end(), h_PHI1.begin());
			//double *raw_PHI1 = thrust::raw_pointer_cast(h_PHI1.data());
			//printMatrix(raw_PHI1, dim3(1, rows));
			//
			//printf("Phi2:\r\n");
			//host_vector<double> h_PHI2(d_PHI2.size());
			//thrust::copy(d_PHI2.begin(), d_PHI2.end(), h_PHI2.begin());
			//double *raw_PHI2 = thrust::raw_pointer_cast(h_PHI2.data());
			//printMatrix(raw_PHI2, dim3(1, rows));
			//
			//printf("D:\r\n");
			//host_vector<double> hv_D(d_D.size());
			//thrust::copy(d_D.begin(), d_D.end(), hv_D.begin());
			//double *raw_D = thrust::raw_pointer_cast(hv_D.data());
			//printMatrix(raw_D, dim3(1, rows));

			//printf("Dt:\r\n");
			//host_vector<double> hv_Dt(d_Dt.size());
			//thrust::copy(d_Dt.begin(), d_Dt.end(), hv_Dt.begin());
			//double *raw_Dt = thrust::raw_pointer_cast(hv_Dt.data());
			//printMatrix(raw_Dt, dim3(1, rows));

			//printf("Dx:\r\n");
			//host_vector<double> hv_Dx(d_Dx.size());
			//thrust::copy(d_Dx.begin(), d_Dx.end(), hv_Dx.begin());
			//double *raw_Dx = thrust::raw_pointer_cast(hv_Dx.data());
			//printMatrix(raw_Dx, dim3(1, rows));

			//printf("Dxx:\r\n");
			//host_vector<double> hv_Dxx(d_Dxx.size());
			//thrust::copy(d_Dxx.begin(), d_Dxx.end(), hv_Dxx.begin());
			//double *raw_Dxx = thrust::raw_pointer_cast(hv_Dxx.data());
			//printMatrix(raw_Dxx, dim3(1, rows));

			//printf("test1:\r\n");
			//host_vector<double> h_test1(d_test1.size());
			//thrust::copy(d_test1.begin(), d_test1.end(), h_test1.begin());
			//double *raw_test1 = thrust::raw_pointer_cast(h_test1.data());
			//printMatrix(raw_test1, dim3(1, rows));

			//printf("test2:\r\n");
			//host_vector<double> h_test2(d_test2.size());
			//thrust::copy(d_test2.begin(), d_test2.end(), h_test2.begin());
			//double *raw_test2 = thrust::raw_pointer_cast(h_test2.data());
			//printMatrix(raw_test2, dim3(1, rows));

			hipDeviceSynchronize();
			//host_vector<double> h_D(rows * rows);
			//thrust::copy(d_D.begin(), d_D.end(), h_D.begin());
			//hipDeviceSynchronize();
			//MatrixXd D = MatrixXd::Zero(rows, rows +1);
			/*Eigen::Map<Eigen::MatrixXd> dataMapD(h_D.data(), rows, rows);
			Matrix<double, 15,15,StorageOptions::DontAlign> D = dataMapD.eval();*/
			//CopyToMatrix(D, h_D.data(), dim3(rows, rows));
			//wcout << printMatrix(D)<< endl;
			double *h_D = (double*)malloc(sizeof(double) * rows * rows);
			double *p_D = d_D.data().get();
			hipError_t e = hipMemcpy(h_D, p_D, sizeof(double) * rows * rows, hipMemcpyKind::hipMemcpyDeviceToHost);
			if (e != hipSuccess)
				printf("hipMemcpy h_D returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * rows);
			Eigen::Map<Eigen::MatrixXd> dataMapD(h_D, rows, rows);
			MatrixXd D = dataMapD.eval();

			hipDeviceSynchronize();
			double *h_Dt = (double*)malloc(sizeof(double) * rows * rows);
			double *p_Dt = d_Dt.data().get();
			e = hipMemcpy(h_Dt, p_Dt, sizeof(double) * rows * rows, hipMemcpyKind::hipMemcpyDeviceToHost);
			if (e != hipSuccess)
				printf("hipMemcpy h_Dt returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * rows);
			Eigen::Map<Eigen::MatrixXd> dataMapDt(h_Dt, rows, rows);
			MatrixXd Dt = dataMapDt.eval();

			hipDeviceSynchronize();
			double *h_Dx = (double*)malloc(sizeof(double) * rows * rows);
			double *p_Dx = d_Dx.data().get();
			e = hipMemcpy(h_Dx, p_Dx, sizeof(double) * rows * rows, hipMemcpyKind::hipMemcpyDeviceToHost);
			if (e != hipSuccess)
				printf("hipMemcpy h_Dx returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * rows);
			Eigen::Map<Eigen::MatrixXd> dataMapDx(h_Dx, rows, rows);
			MatrixXd Dx = dataMapDx.eval();

			hipDeviceSynchronize();
			double *h_Dxx = (double*)malloc(sizeof(double) * rows * rows);
			double *p_Dxx = d_Dxx.data().get();
			e = hipMemcpy(h_Dxx, p_Dxx, sizeof(double) * rows * rows, hipMemcpyKind::hipMemcpyDeviceToHost);
			if (e != hipSuccess)
				printf("hipMemcpy h_Dxx returned error %s (code %d), line(%d) when copying%i\n",
					hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * rows);
			Eigen::Map<Eigen::MatrixXd> dataMapDxx(h_Dxx, rows, rows);
			MatrixXd Dxx = dataMapDxx.eval();

			return{ D, Dt, Dx, Dxx };
		}
	}
}