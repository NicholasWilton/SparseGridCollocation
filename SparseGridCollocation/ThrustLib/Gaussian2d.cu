#include "hip/hip_runtime.h"
#include "Gaussian2d.h"

using Eigen::MatrixXd;
using Eigen::VectorXd;
using namespace std;
using namespace Eigen;
using namespace thrust;



void printMatrix(const double *matrix, dim3 dimMatrix)
{
	int mSize = sizeof(matrix);

	printf("printing matrix data=");
	for (int x = 0; x < dimMatrix.x * dimMatrix.y; x++)
		printf("%f,", matrix[x]);
	printf("\r\n");
	printf("rows=%i cols=%i\r\n", dimMatrix.y, dimMatrix.x);

	for (int y = 0; y < dimMatrix.y; y++)
	{
		for (int x = 0; x < dimMatrix.x; x++)
		{
			int idx = (x * dimMatrix.y) + y;
			printf("%.16f ", matrix[idx]);
		}
		printf("\r\n");
	}
}

wstring printMatrix(MatrixXd m)
{
	int cols = m.cols();
	int rows = m.rows();

	wstringstream ss;
	ss << setprecision(25);
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			double d = m(i, j);
			ss << d << "\t";

		}
		ss << "\r\n";
	}

	return ss.str();
}

struct phi_functor
{
	const double a;
	const double cn;
	const double c;
	phi_functor(double _a, double _cn, double _c) : a(_a), cn(_cn), c(_c) {}
	__device__
		double operator()(const double &TP)
	{
		double a1 = a * (TP - cn);
		double b1 = -(a1 * a1) / (c*c);
		double e1 = expm1(b1) + 1;
		return e1;
	}
};

struct vectorAddScalar_functor
{
	const double a;
	
	vectorAddScalar_functor(double _a) : a(_a) {}
	__device__
		double operator()(const double &TP)
	{
		return TP + a;
	}
};

struct scalarVectorDifference_functor
{
	const double a;
	const double cn;
	
	scalarVectorDifference_functor(double _a, double _cn) : a(_a), cn(_cn){}
	__device__
		double operator()(const double &TP)
	{
		return a * (TP - cn);
	}
};

struct vectorScalarDifference_functor
{
	const double a;
	
	vectorScalarDifference_functor(double _a) : a(_a) {}
	__device__
		double operator()(const double &TP)
	{
		return TP - a;
	}
};

struct vectorScalarAddition_functor
{
	const double a;

	vectorScalarAddition_functor(double _a) : a(_a) {}
	__device__
		double operator()(const double &TP)
	{
		return TP + a;
	}
};

struct vectorScalarMultiply_functor
{
	const double a;

	vectorScalarMultiply_functor(double _a) : a(_a) {}
	__device__
		double operator()(const double &TP)
	{
		return TP * a;
	}
};

void CopyToMatrix(MatrixXd &m, double* buffer, dim3 size)
{
	int ptr = 0;
	for (int i = 0; i < size.x; i++)
		for (int j = 0; j < size.y; j++)
			m(i, j) = buffer[ptr];
}

vector<MatrixXd> ThrustLib::Gaussian::Gaussian2d(const MatrixXd & TP, const MatrixXd & CN, const MatrixXd & A, const MatrixXd & C)
{
	int rows = TP.rows();
	const double *h_tp0 = TP.col(0).data();
	const double *h_tp1 = TP.col(1).data();
	const double *h_cn = CN.data();
	const double *h_a = A.data();
	const double *h_c = C.data();
	//wcout << printMatrix(TP.col(0)) << endl;
	//wcout << printMatrix(TP.col(1)) << endl;
	
	device_vector<double> d_tp0(h_tp0, h_tp0 + (TP.rows()));
	device_vector<double> d_tp1(h_tp1, h_tp1 + (TP.rows()));
	device_vector<double> d_cn(h_cn, h_cn + (CN.rows() * CN.cols()));
	device_vector<double> d_a(h_a, h_a + (A.rows() * A.cols()));
	device_vector<double> d_c(h_c, h_c + (C.rows() * C.cols()));

	device_vector<double> d_PHI1(rows * rows);
	device_vector<double> d_PHI2(rows * rows);
	device_vector<double> d_D(rows * rows);
	device_vector<double> d_Dt(rows * rows);
	device_vector<double> d_Dx(rows * rows);
	device_vector<double> d_Dxx(rows * rows);

	device_vector<double> d_test1(rows * rows);
	device_vector<double> d_test2(rows * rows);
	
	for (int i = 0; i < rows; i++)
	{
		//if (i == 0)
		//{
		//	d_test1.insert(d_test1.begin(), d_tp0.begin(), d_tp0.end());
		//	d_test2.insert(d_test2.begin(), d_tp1.begin(), d_tp1.end());
		//}
		//Calculate Phi1 & Phi2
		thrust::device_vector<double> phi1(rows);
		thrust::transform(d_tp0.begin(), d_tp0.end(), phi1.begin(), phi_functor(A(0,0), CN(i,0), C(0,0)));
		d_PHI1.insert(d_PHI1.begin(), phi1.begin(), phi1.end());

		thrust::device_vector<double> phi2(rows);
		thrust::transform(d_tp1.begin(), d_tp1.end(), phi2.begin(), phi_functor(A(0, 1), CN(i, 1), C(0, 1)));
		d_PHI2.insert(d_PHI2.begin(), phi2.begin(), phi2.end());

		//Calculate D
		thrust::device_vector<double> d(rows);
		thrust::transform(phi1.begin(), phi1.end(), phi2.begin(), d.begin(), thrust::multiplies<double>());
		d_D.insert(d_D.begin(), d.begin(), d.end());

		//Calculate Dt
		thrust::device_vector<double> a1(rows);
		double scalarDt = -2 * (A(0, 0) / C(0, 0)) * (A(0, 0) / C(0, 0));
		thrust::transform(d_tp0.begin(), d_tp0.end(), a1.begin(), scalarVectorDifference_functor(scalarDt, CN(i, 0)));
		
		thrust::device_vector<double> b1(rows);
		thrust::transform(phi1.begin(), phi1.end(), a1.begin(), b1.begin(), thrust::multiplies<double>());

		thrust::device_vector<double> dt(rows);
		thrust::transform(phi2.begin(), phi2.end(), b1.begin(), dt.begin(), thrust::multiplies<double>());
		d_Dt.insert(d_Dt.begin(), dt.begin(), dt.end());

		//Calculate Dx
		double scalarDx = -2 * ((A(0,1) / C(0,1)) * (A(0, 1) / C(0, 1)));
		thrust::device_vector<double> a2(rows);
		thrust::transform(d_tp1.begin(), d_tp1.end(), a2.begin(), scalarVectorDifference_functor(scalarDx, CN(i, 1)));

		thrust::device_vector<double> b2(rows);
		thrust::transform(phi1.begin(), phi1.end(), a2.begin(), b2.begin(), thrust::multiplies<double>());

		thrust::device_vector<double> c2(rows);
		thrust::transform(phi2.begin(), phi2.end(), b2.begin(), c2.begin(), thrust::multiplies<double>());

		thrust::device_vector<double> dx(rows);
		thrust::transform(d_tp1.begin(), d_tp1.end(), c2.begin(), dx.begin(), thrust::multiplies<double>());
		d_Dx.insert(d_Dx.begin(), dx.begin(), dx.end());

		//Calculate Dxx
		double sA = A(0,1) * A(0, 1);
		double qA = sA * sA;
		double sC = C(0, 1) * C(0, 1);
		double qC = sC * sC;

		thrust::device_vector<double> dTpCn(rows);
		thrust::transform(d_tp1.begin(), d_tp1.end(), dTpCn.begin(), vectorScalarDifference_functor(CN(0,1)));
		
		thrust::device_vector<double> sDTpCn(rows);
		thrust::transform(dTpCn.begin(), dTpCn.end(), dTpCn.begin(), sDTpCn.begin(), thrust::multiplies<double>());
		
		thrust::device_vector<double> a3(rows);
		double scalarDxx1 = 4 * qA / qC;
		
		thrust::transform(sDTpCn.begin(), sDTpCn.end(), a3.begin(), vectorScalarMultiply_functor(scalarDxx1));
		d_test1.insert(d_test1.begin(), a3.begin(), a3.end());
		thrust::device_vector<double> b3(rows);
		double scalarDxx2 = -2 * sA / sC;
		
		thrust::transform(a3.begin(), a3.end(), b3.begin(), vectorAddScalar_functor(scalarDxx2));
		
		thrust::device_vector<double> c3(rows);
		thrust::transform(d.begin(), d.end(), b3.begin(), c3.begin(), thrust::multiplies<double>());
		
		thrust::device_vector<double> sTpCol1(rows);
		thrust::transform(d_tp1.begin(), d_tp1.end(), d_tp1.begin(), sTpCol1.begin(), thrust::multiplies<double>());
		
		thrust::device_vector<double> dxx(rows);
		//thrust::device_vector<double> rsTpCol(rows);
		//thrust::copy(sTpCol1.rbegin(), sTpCol1.rend(), rsTpCol.begin());
		thrust::transform(c3.begin(), c3.end(), sTpCol1.begin(), dxx.begin(), thrust::multiplies<double>());
		d_Dxx.insert(d_Dxx.begin(), dxx.begin(), dxx.end());

		if (i == 1)
		{
			//d_test1.insert(d_test1.begin(), d_tp1.begin(), d_tp1.end());
			d_test2.insert(d_test2.begin(), d_Dxx.begin(), d_Dxx.end());
		}
	}

	//printf("Phi1:\r\n");
	//host_vector<double> h_PHI1(d_PHI1.size());
	//thrust::copy(d_PHI1.begin(), d_PHI1.end(), h_PHI1.begin());
	//double *raw_PHI1 = thrust::raw_pointer_cast(h_PHI1.data());
	//printMatrix(raw_PHI1, dim3(1, rows));
	//
	//printf("Phi2:\r\n");
	//host_vector<double> h_PHI2(d_PHI2.size());
	//thrust::copy(d_PHI2.begin(), d_PHI2.end(), h_PHI2.begin());
	//double *raw_PHI2 = thrust::raw_pointer_cast(h_PHI2.data());
	//printMatrix(raw_PHI2, dim3(1, rows));
	//
	//printf("D:\r\n");
	//host_vector<double> h_D(d_D.size());
	//thrust::copy(d_D.begin(), d_D.end(), h_D.begin());
	//double *raw_D = thrust::raw_pointer_cast(h_D.data());
	//printMatrix(raw_D, dim3(1, rows));

	//printf("Dt:\r\n");
	//host_vector<double> h_Dt(d_Dt.size());
	//thrust::copy(d_Dt.begin(), d_Dt.end(), h_Dt.begin());
	//double *raw_Dt = thrust::raw_pointer_cast(h_Dt.data());
	//printMatrix(raw_Dt, dim3(1, rows));

	//printf("Dx:\r\n");
	//host_vector<double> h_Dx(d_Dx.size());
	//thrust::copy(d_Dx.begin(), d_Dx.end(), h_Dx.begin());
	//double *raw_Dx = thrust::raw_pointer_cast(h_Dx.data());
	//printMatrix(raw_Dx, dim3(1, rows));

	//printf("Dxx:\r\n");
	//host_vector<double> h_Dxx(d_Dxx.size());
	//thrust::copy(d_Dxx.begin(), d_Dxx.end(), h_Dxx.begin());
	//double *raw_Dxx = thrust::raw_pointer_cast(h_Dxx.data());
	//printMatrix(raw_Dxx, dim3(1, rows));

	//printf("test1:\r\n");
	//host_vector<double> h_test1(d_test1.size());
	//thrust::copy(d_test1.begin(), d_test1.end(), h_test1.begin());
	//double *raw_test1 = thrust::raw_pointer_cast(h_test1.data());
	//printMatrix(raw_test1, dim3(1, rows));

	//printf("test2:\r\n");
	//host_vector<double> h_test2(d_test2.size());
	//thrust::copy(d_test2.begin(), d_test2.end(), h_test2.begin());
	//double *raw_test2 = thrust::raw_pointer_cast(h_test2.data());
	//printMatrix(raw_test2, dim3(1, rows));
	
	hipDeviceSynchronize();
	//host_vector<double> h_D(rows * rows);
	//thrust::copy(d_D.begin(), d_D.end(), h_D.begin());
	//hipDeviceSynchronize();
	//MatrixXd D = MatrixXd::Zero(rows, rows +1);
	/*Eigen::Map<Eigen::MatrixXd> dataMapD(h_D.data(), rows, rows);
	Matrix<double, 15,15,StorageOptions::DontAlign> D = dataMapD.eval();*/
	//CopyToMatrix(D, h_D.data(), dim3(rows, rows));
	//wcout << printMatrix(D)<< endl;
	double *h_D = (double*)malloc(sizeof(double) * rows * rows);
	double *p_D = d_D.data().get();
	hipError_t e = hipMemcpy(h_D, p_D, sizeof(double) * rows * rows, hipMemcpyKind::hipMemcpyDeviceToHost);
	if (e != hipSuccess)
		printf("hipMemcpy h_D returned error %s (code %d), line(%d) when copying%i\n", 
			hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * rows);
	Eigen::Map<Eigen::MatrixXd> dataMapD(h_D, rows, rows);
	MatrixXd D = dataMapD.eval();

	hipDeviceSynchronize();
	double *h_Dt = (double*)malloc(sizeof(double) * rows * rows);
	double *p_Dt = d_Dt.data().get();
	e = hipMemcpy(h_Dt, p_Dt, sizeof(double) * rows * rows, hipMemcpyKind::hipMemcpyDeviceToHost);
	if (e != hipSuccess)
		printf("hipMemcpy h_Dt returned error %s (code %d), line(%d) when copying%i\n",
			hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * rows);
	Eigen::Map<Eigen::MatrixXd> dataMapDt(h_Dt, rows, rows);
	MatrixXd Dt = dataMapDt.eval();
	
	hipDeviceSynchronize();
	double *h_Dx = (double*)malloc(sizeof(double) * rows * rows);
	double *p_Dx = d_Dx.data().get();
	e = hipMemcpy(h_Dx, p_Dx, sizeof(double) * rows * rows, hipMemcpyKind::hipMemcpyDeviceToHost);
	if (e != hipSuccess)
		printf("hipMemcpy h_Dx returned error %s (code %d), line(%d) when copying%i\n",
			hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * rows);
	Eigen::Map<Eigen::MatrixXd> dataMapDx(h_Dx, rows, rows);
	MatrixXd Dx = dataMapDx.eval();

	hipDeviceSynchronize();
	double *h_Dxx = (double*)malloc(sizeof(double) * rows * rows);
	double *p_Dxx = d_Dxx.data().get();
	e = hipMemcpy(h_Dxx, p_Dxx, sizeof(double) * rows * rows, hipMemcpyKind::hipMemcpyDeviceToHost);
	if (e != hipSuccess)
		printf("hipMemcpy h_Dxx returned error %s (code %d), line(%d) when copying%i\n",
			hipGetErrorString(e), e, __LINE__, sizeof(double) * rows * rows);
	Eigen::Map<Eigen::MatrixXd> dataMapDxx(h_Dxx, rows, rows);
	MatrixXd Dxx = dataMapDxx.eval();

	return {D, Dt, Dx, Dxx};
}

MatrixXd GetTX7()
{
	MatrixXd TX1(325, 2);
	TX1(0, 0) = 0;
	TX1(1, 0) = 0;
	TX1(2, 0) = 0;
	TX1(3, 0) = 0;
	TX1(4, 0) = 0;
	TX1(5, 0) = 0;
	TX1(6, 0) = 0;
	TX1(7, 0) = 0;
	TX1(8, 0) = 0;
	TX1(9, 0) = 0;
	TX1(10, 0) = 0;
	TX1(11, 0) = 0;
	TX1(12, 0) = 0;
	TX1(13, 0) = 0;
	TX1(14, 0) = 0;
	TX1(15, 0) = 0;
	TX1(16, 0) = 0;
	TX1(17, 0) = 0;
	TX1(18, 0) = 0;
	TX1(19, 0) = 0;
	TX1(20, 0) = 0;
	TX1(21, 0) = 0;
	TX1(22, 0) = 0;
	TX1(23, 0) = 0;
	TX1(24, 0) = 0;
	TX1(25, 0) = 0;
	TX1(26, 0) = 0;
	TX1(27, 0) = 0;
	TX1(28, 0) = 0;
	TX1(29, 0) = 0;
	TX1(30, 0) = 0;
	TX1(31, 0) = 0;
	TX1(32, 0) = 0;
	TX1(33, 0) = 0;
	TX1(34, 0) = 0;
	TX1(35, 0) = 0;
	TX1(36, 0) = 0;
	TX1(37, 0) = 0;
	TX1(38, 0) = 0;
	TX1(39, 0) = 0;
	TX1(40, 0) = 0;
	TX1(41, 0) = 0;
	TX1(42, 0) = 0;
	TX1(43, 0) = 0;
	TX1(44, 0) = 0;
	TX1(45, 0) = 0;
	TX1(46, 0) = 0;
	TX1(47, 0) = 0;
	TX1(48, 0) = 0;
	TX1(49, 0) = 0;
	TX1(50, 0) = 0;
	TX1(51, 0) = 0;
	TX1(52, 0) = 0;
	TX1(53, 0) = 0;
	TX1(54, 0) = 0;
	TX1(55, 0) = 0;
	TX1(56, 0) = 0;
	TX1(57, 0) = 0;
	TX1(58, 0) = 0;
	TX1(59, 0) = 0;
	TX1(60, 0) = 0;
	TX1(61, 0) = 0;
	TX1(62, 0) = 0;
	TX1(63, 0) = 0;
	TX1(64, 0) = 0;
	TX1(65, 0) = 0.21625;
	TX1(66, 0) = 0.21625;
	TX1(67, 0) = 0.21625;
	TX1(68, 0) = 0.21625;
	TX1(69, 0) = 0.21625;
	TX1(70, 0) = 0.21625;
	TX1(71, 0) = 0.21625;
	TX1(72, 0) = 0.21625;
	TX1(73, 0) = 0.21625;
	TX1(74, 0) = 0.21625;
	TX1(75, 0) = 0.21625;
	TX1(76, 0) = 0.21625;
	TX1(77, 0) = 0.21625;
	TX1(78, 0) = 0.21625;
	TX1(79, 0) = 0.21625;
	TX1(80, 0) = 0.21625;
	TX1(81, 0) = 0.21625;
	TX1(82, 0) = 0.21625;
	TX1(83, 0) = 0.21625;
	TX1(84, 0) = 0.21625;
	TX1(85, 0) = 0.21625;
	TX1(86, 0) = 0.21625;
	TX1(87, 0) = 0.21625;
	TX1(88, 0) = 0.21625;
	TX1(89, 0) = 0.21625;
	TX1(90, 0) = 0.21625;
	TX1(91, 0) = 0.21625;
	TX1(92, 0) = 0.21625;
	TX1(93, 0) = 0.21625;
	TX1(94, 0) = 0.21625;
	TX1(95, 0) = 0.21625;
	TX1(96, 0) = 0.21625;
	TX1(97, 0) = 0.21625;
	TX1(98, 0) = 0.21625;
	TX1(99, 0) = 0.21625;
	TX1(100, 0) = 0.21625;
	TX1(101, 0) = 0.21625;
	TX1(102, 0) = 0.21625;
	TX1(103, 0) = 0.21625;
	TX1(104, 0) = 0.21625;
	TX1(105, 0) = 0.21625;
	TX1(106, 0) = 0.21625;
	TX1(107, 0) = 0.21625;
	TX1(108, 0) = 0.21625;
	TX1(109, 0) = 0.21625;
	TX1(110, 0) = 0.21625;
	TX1(111, 0) = 0.21625;
	TX1(112, 0) = 0.21625;
	TX1(113, 0) = 0.21625;
	TX1(114, 0) = 0.21625;
	TX1(115, 0) = 0.21625;
	TX1(116, 0) = 0.21625;
	TX1(117, 0) = 0.21625;
	TX1(118, 0) = 0.21625;
	TX1(119, 0) = 0.21625;
	TX1(120, 0) = 0.21625;
	TX1(121, 0) = 0.21625;
	TX1(122, 0) = 0.21625;
	TX1(123, 0) = 0.21625;
	TX1(124, 0) = 0.21625;
	TX1(125, 0) = 0.21625;
	TX1(126, 0) = 0.21625;
	TX1(127, 0) = 0.21625;
	TX1(128, 0) = 0.21625;
	TX1(129, 0) = 0.21625;
	TX1(130, 0) = 0.432499999999999;
	TX1(131, 0) = 0.432499999999999;
	TX1(132, 0) = 0.432499999999999;
	TX1(133, 0) = 0.432499999999999;
	TX1(134, 0) = 0.432499999999999;
	TX1(135, 0) = 0.432499999999999;
	TX1(136, 0) = 0.432499999999999;
	TX1(137, 0) = 0.432499999999999;
	TX1(138, 0) = 0.432499999999999;
	TX1(139, 0) = 0.432499999999999;
	TX1(140, 0) = 0.432499999999999;
	TX1(141, 0) = 0.432499999999999;
	TX1(142, 0) = 0.432499999999999;
	TX1(143, 0) = 0.432499999999999;
	TX1(144, 0) = 0.432499999999999;
	TX1(145, 0) = 0.432499999999999;
	TX1(146, 0) = 0.432499999999999;
	TX1(147, 0) = 0.432499999999999;
	TX1(148, 0) = 0.432499999999999;
	TX1(149, 0) = 0.432499999999999;
	TX1(150, 0) = 0.432499999999999;
	TX1(151, 0) = 0.432499999999999;
	TX1(152, 0) = 0.432499999999999;
	TX1(153, 0) = 0.432499999999999;
	TX1(154, 0) = 0.432499999999999;
	TX1(155, 0) = 0.432499999999999;
	TX1(156, 0) = 0.432499999999999;
	TX1(157, 0) = 0.432499999999999;
	TX1(158, 0) = 0.432499999999999;
	TX1(159, 0) = 0.432499999999999;
	TX1(160, 0) = 0.432499999999999;
	TX1(161, 0) = 0.432499999999999;
	TX1(162, 0) = 0.432499999999999;
	TX1(163, 0) = 0.432499999999999;
	TX1(164, 0) = 0.432499999999999;
	TX1(165, 0) = 0.432499999999999;
	TX1(166, 0) = 0.432499999999999;
	TX1(167, 0) = 0.432499999999999;
	TX1(168, 0) = 0.432499999999999;
	TX1(169, 0) = 0.432499999999999;
	TX1(170, 0) = 0.432499999999999;
	TX1(171, 0) = 0.432499999999999;
	TX1(172, 0) = 0.432499999999999;
	TX1(173, 0) = 0.432499999999999;
	TX1(174, 0) = 0.432499999999999;
	TX1(175, 0) = 0.432499999999999;
	TX1(176, 0) = 0.432499999999999;
	TX1(177, 0) = 0.432499999999999;
	TX1(178, 0) = 0.432499999999999;
	TX1(179, 0) = 0.432499999999999;
	TX1(180, 0) = 0.432499999999999;
	TX1(181, 0) = 0.432499999999999;
	TX1(182, 0) = 0.432499999999999;
	TX1(183, 0) = 0.432499999999999;
	TX1(184, 0) = 0.432499999999999;
	TX1(185, 0) = 0.432499999999999;
	TX1(186, 0) = 0.432499999999999;
	TX1(187, 0) = 0.432499999999999;
	TX1(188, 0) = 0.432499999999999;
	TX1(189, 0) = 0.432499999999999;
	TX1(190, 0) = 0.432499999999999;
	TX1(191, 0) = 0.432499999999999;
	TX1(192, 0) = 0.432499999999999;
	TX1(193, 0) = 0.432499999999999;
	TX1(194, 0) = 0.432499999999999;
	TX1(195, 0) = 0.648749999999999;
	TX1(196, 0) = 0.648749999999999;
	TX1(197, 0) = 0.648749999999999;
	TX1(198, 0) = 0.648749999999999;
	TX1(199, 0) = 0.648749999999999;
	TX1(200, 0) = 0.648749999999999;
	TX1(201, 0) = 0.648749999999999;
	TX1(202, 0) = 0.648749999999999;
	TX1(203, 0) = 0.648749999999999;
	TX1(204, 0) = 0.648749999999999;
	TX1(205, 0) = 0.648749999999999;
	TX1(206, 0) = 0.648749999999999;
	TX1(207, 0) = 0.648749999999999;
	TX1(208, 0) = 0.648749999999999;
	TX1(209, 0) = 0.648749999999999;
	TX1(210, 0) = 0.648749999999999;
	TX1(211, 0) = 0.648749999999999;
	TX1(212, 0) = 0.648749999999999;
	TX1(213, 0) = 0.648749999999999;
	TX1(214, 0) = 0.648749999999999;
	TX1(215, 0) = 0.648749999999999;
	TX1(216, 0) = 0.648749999999999;
	TX1(217, 0) = 0.648749999999999;
	TX1(218, 0) = 0.648749999999999;
	TX1(219, 0) = 0.648749999999999;
	TX1(220, 0) = 0.648749999999999;
	TX1(221, 0) = 0.648749999999999;
	TX1(222, 0) = 0.648749999999999;
	TX1(223, 0) = 0.648749999999999;
	TX1(224, 0) = 0.648749999999999;
	TX1(225, 0) = 0.648749999999999;
	TX1(226, 0) = 0.648749999999999;
	TX1(227, 0) = 0.648749999999999;
	TX1(228, 0) = 0.648749999999999;
	TX1(229, 0) = 0.648749999999999;
	TX1(230, 0) = 0.648749999999999;
	TX1(231, 0) = 0.648749999999999;
	TX1(232, 0) = 0.648749999999999;
	TX1(233, 0) = 0.648749999999999;
	TX1(234, 0) = 0.648749999999999;
	TX1(235, 0) = 0.648749999999999;
	TX1(236, 0) = 0.648749999999999;
	TX1(237, 0) = 0.648749999999999;
	TX1(238, 0) = 0.648749999999999;
	TX1(239, 0) = 0.648749999999999;
	TX1(240, 0) = 0.648749999999999;
	TX1(241, 0) = 0.648749999999999;
	TX1(242, 0) = 0.648749999999999;
	TX1(243, 0) = 0.648749999999999;
	TX1(244, 0) = 0.648749999999999;
	TX1(245, 0) = 0.648749999999999;
	TX1(246, 0) = 0.648749999999999;
	TX1(247, 0) = 0.648749999999999;
	TX1(248, 0) = 0.648749999999999;
	TX1(249, 0) = 0.648749999999999;
	TX1(250, 0) = 0.648749999999999;
	TX1(251, 0) = 0.648749999999999;
	TX1(252, 0) = 0.648749999999999;
	TX1(253, 0) = 0.648749999999999;
	TX1(254, 0) = 0.648749999999999;
	TX1(255, 0) = 0.648749999999999;
	TX1(256, 0) = 0.648749999999999;
	TX1(257, 0) = 0.648749999999999;
	TX1(258, 0) = 0.648749999999999;
	TX1(259, 0) = 0.648749999999999;
	TX1(260, 0) = 0.864999999999999;
	TX1(261, 0) = 0.864999999999999;
	TX1(262, 0) = 0.864999999999999;
	TX1(263, 0) = 0.864999999999999;
	TX1(264, 0) = 0.864999999999999;
	TX1(265, 0) = 0.864999999999999;
	TX1(266, 0) = 0.864999999999999;
	TX1(267, 0) = 0.864999999999999;
	TX1(268, 0) = 0.864999999999999;
	TX1(269, 0) = 0.864999999999999;
	TX1(270, 0) = 0.864999999999999;
	TX1(271, 0) = 0.864999999999999;
	TX1(272, 0) = 0.864999999999999;
	TX1(273, 0) = 0.864999999999999;
	TX1(274, 0) = 0.864999999999999;
	TX1(275, 0) = 0.864999999999999;
	TX1(276, 0) = 0.864999999999999;
	TX1(277, 0) = 0.864999999999999;
	TX1(278, 0) = 0.864999999999999;
	TX1(279, 0) = 0.864999999999999;
	TX1(280, 0) = 0.864999999999999;
	TX1(281, 0) = 0.864999999999999;
	TX1(282, 0) = 0.864999999999999;
	TX1(283, 0) = 0.864999999999999;
	TX1(284, 0) = 0.864999999999999;
	TX1(285, 0) = 0.864999999999999;
	TX1(286, 0) = 0.864999999999999;
	TX1(287, 0) = 0.864999999999999;
	TX1(288, 0) = 0.864999999999999;
	TX1(289, 0) = 0.864999999999999;
	TX1(290, 0) = 0.864999999999999;
	TX1(291, 0) = 0.864999999999999;
	TX1(292, 0) = 0.864999999999999;
	TX1(293, 0) = 0.864999999999999;
	TX1(294, 0) = 0.864999999999999;
	TX1(295, 0) = 0.864999999999999;
	TX1(296, 0) = 0.864999999999999;
	TX1(297, 0) = 0.864999999999999;
	TX1(298, 0) = 0.864999999999999;
	TX1(299, 0) = 0.864999999999999;
	TX1(300, 0) = 0.864999999999999;
	TX1(301, 0) = 0.864999999999999;
	TX1(302, 0) = 0.864999999999999;
	TX1(303, 0) = 0.864999999999999;
	TX1(304, 0) = 0.864999999999999;
	TX1(305, 0) = 0.864999999999999;
	TX1(306, 0) = 0.864999999999999;
	TX1(307, 0) = 0.864999999999999;
	TX1(308, 0) = 0.864999999999999;
	TX1(309, 0) = 0.864999999999999;
	TX1(310, 0) = 0.864999999999999;
	TX1(311, 0) = 0.864999999999999;
	TX1(312, 0) = 0.864999999999999;
	TX1(313, 0) = 0.864999999999999;
	TX1(314, 0) = 0.864999999999999;
	TX1(315, 0) = 0.864999999999999;
	TX1(316, 0) = 0.864999999999999;
	TX1(317, 0) = 0.864999999999999;
	TX1(318, 0) = 0.864999999999999;
	TX1(319, 0) = 0.864999999999999;
	TX1(320, 0) = 0.864999999999999;
	TX1(321, 0) = 0.864999999999999;
	TX1(322, 0) = 0.864999999999999;
	TX1(323, 0) = 0.864999999999999;
	TX1(324, 0) = 0.864999999999999;
	TX1(0, 1) = 0;
	TX1(1, 1) = 4.6875;
	TX1(2, 1) = 9.375;
	TX1(3, 1) = 14.0625;
	TX1(4, 1) = 18.75;
	TX1(5, 1) = 23.4375;
	TX1(6, 1) = 28.125;
	TX1(7, 1) = 32.8125;
	TX1(8, 1) = 37.5;
	TX1(9, 1) = 42.1875;
	TX1(10, 1) = 46.875;
	TX1(11, 1) = 51.5625;
	TX1(12, 1) = 56.25;
	TX1(13, 1) = 60.9375;
	TX1(14, 1) = 65.625;
	TX1(15, 1) = 70.3125;
	TX1(16, 1) = 75;
	TX1(17, 1) = 79.6875;
	TX1(18, 1) = 84.375;
	TX1(19, 1) = 89.0625;
	TX1(20, 1) = 93.75;
	TX1(21, 1) = 98.4375;
	TX1(22, 1) = 103.125;
	TX1(23, 1) = 107.8125;
	TX1(24, 1) = 112.5;
	TX1(25, 1) = 117.1875;
	TX1(26, 1) = 121.875;
	TX1(27, 1) = 126.5625;
	TX1(28, 1) = 131.25;
	TX1(29, 1) = 135.9375;
	TX1(30, 1) = 140.625;
	TX1(31, 1) = 145.3125;
	TX1(32, 1) = 150;
	TX1(33, 1) = 154.6875;
	TX1(34, 1) = 159.375;
	TX1(35, 1) = 164.0625;
	TX1(36, 1) = 168.75;
	TX1(37, 1) = 173.4375;
	TX1(38, 1) = 178.125;
	TX1(39, 1) = 182.8125;
	TX1(40, 1) = 187.5;
	TX1(41, 1) = 192.1875;
	TX1(42, 1) = 196.875;
	TX1(43, 1) = 201.5625;
	TX1(44, 1) = 206.25;
	TX1(45, 1) = 210.9375;
	TX1(46, 1) = 215.625;
	TX1(47, 1) = 220.3125;
	TX1(48, 1) = 225;
	TX1(49, 1) = 229.6875;
	TX1(50, 1) = 234.375;
	TX1(51, 1) = 239.0625;
	TX1(52, 1) = 243.75;
	TX1(53, 1) = 248.4375;
	TX1(54, 1) = 253.125;
	TX1(55, 1) = 257.8125;
	TX1(56, 1) = 262.5;
	TX1(57, 1) = 267.1875;
	TX1(58, 1) = 271.875;
	TX1(59, 1) = 276.5625;
	TX1(60, 1) = 281.25;
	TX1(61, 1) = 285.9375;
	TX1(62, 1) = 290.625;
	TX1(63, 1) = 295.3125;
	TX1(64, 1) = 300;
	TX1(65, 1) = 0;
	TX1(66, 1) = 4.6875;
	TX1(67, 1) = 9.375;
	TX1(68, 1) = 14.0625;
	TX1(69, 1) = 18.75;
	TX1(70, 1) = 23.4375;
	TX1(71, 1) = 28.125;
	TX1(72, 1) = 32.8125;
	TX1(73, 1) = 37.5;
	TX1(74, 1) = 42.1875;
	TX1(75, 1) = 46.875;
	TX1(76, 1) = 51.5625;
	TX1(77, 1) = 56.25;
	TX1(78, 1) = 60.9375;
	TX1(79, 1) = 65.625;
	TX1(80, 1) = 70.3125;
	TX1(81, 1) = 75;
	TX1(82, 1) = 79.6875;
	TX1(83, 1) = 84.375;
	TX1(84, 1) = 89.0625;
	TX1(85, 1) = 93.75;
	TX1(86, 1) = 98.4375;
	TX1(87, 1) = 103.125;
	TX1(88, 1) = 107.8125;
	TX1(89, 1) = 112.5;
	TX1(90, 1) = 117.1875;
	TX1(91, 1) = 121.875;
	TX1(92, 1) = 126.5625;
	TX1(93, 1) = 131.25;
	TX1(94, 1) = 135.9375;
	TX1(95, 1) = 140.625;
	TX1(96, 1) = 145.3125;
	TX1(97, 1) = 150;
	TX1(98, 1) = 154.6875;
	TX1(99, 1) = 159.375;
	TX1(100, 1) = 164.0625;
	TX1(101, 1) = 168.75;
	TX1(102, 1) = 173.4375;
	TX1(103, 1) = 178.125;
	TX1(104, 1) = 182.8125;
	TX1(105, 1) = 187.5;
	TX1(106, 1) = 192.1875;
	TX1(107, 1) = 196.875;
	TX1(108, 1) = 201.5625;
	TX1(109, 1) = 206.25;
	TX1(110, 1) = 210.9375;
	TX1(111, 1) = 215.625;
	TX1(112, 1) = 220.3125;
	TX1(113, 1) = 225;
	TX1(114, 1) = 229.6875;
	TX1(115, 1) = 234.375;
	TX1(116, 1) = 239.0625;
	TX1(117, 1) = 243.75;
	TX1(118, 1) = 248.4375;
	TX1(119, 1) = 253.125;
	TX1(120, 1) = 257.8125;
	TX1(121, 1) = 262.5;
	TX1(122, 1) = 267.1875;
	TX1(123, 1) = 271.875;
	TX1(124, 1) = 276.5625;
	TX1(125, 1) = 281.25;
	TX1(126, 1) = 285.9375;
	TX1(127, 1) = 290.625;
	TX1(128, 1) = 295.3125;
	TX1(129, 1) = 300;
	TX1(130, 1) = 0;
	TX1(131, 1) = 4.6875;
	TX1(132, 1) = 9.375;
	TX1(133, 1) = 14.0625;
	TX1(134, 1) = 18.75;
	TX1(135, 1) = 23.4375;
	TX1(136, 1) = 28.125;
	TX1(137, 1) = 32.8125;
	TX1(138, 1) = 37.5;
	TX1(139, 1) = 42.1875;
	TX1(140, 1) = 46.875;
	TX1(141, 1) = 51.5625;
	TX1(142, 1) = 56.25;
	TX1(143, 1) = 60.9375;
	TX1(144, 1) = 65.625;
	TX1(145, 1) = 70.3125;
	TX1(146, 1) = 75;
	TX1(147, 1) = 79.6875;
	TX1(148, 1) = 84.375;
	TX1(149, 1) = 89.0625;
	TX1(150, 1) = 93.75;
	TX1(151, 1) = 98.4375;
	TX1(152, 1) = 103.125;
	TX1(153, 1) = 107.8125;
	TX1(154, 1) = 112.5;
	TX1(155, 1) = 117.1875;
	TX1(156, 1) = 121.875;
	TX1(157, 1) = 126.5625;
	TX1(158, 1) = 131.25;
	TX1(159, 1) = 135.9375;
	TX1(160, 1) = 140.625;
	TX1(161, 1) = 145.3125;
	TX1(162, 1) = 150;
	TX1(163, 1) = 154.6875;
	TX1(164, 1) = 159.375;
	TX1(165, 1) = 164.0625;
	TX1(166, 1) = 168.75;
	TX1(167, 1) = 173.4375;
	TX1(168, 1) = 178.125;
	TX1(169, 1) = 182.8125;
	TX1(170, 1) = 187.5;
	TX1(171, 1) = 192.1875;
	TX1(172, 1) = 196.875;
	TX1(173, 1) = 201.5625;
	TX1(174, 1) = 206.25;
	TX1(175, 1) = 210.9375;
	TX1(176, 1) = 215.625;
	TX1(177, 1) = 220.3125;
	TX1(178, 1) = 225;
	TX1(179, 1) = 229.6875;
	TX1(180, 1) = 234.375;
	TX1(181, 1) = 239.0625;
	TX1(182, 1) = 243.75;
	TX1(183, 1) = 248.4375;
	TX1(184, 1) = 253.125;
	TX1(185, 1) = 257.8125;
	TX1(186, 1) = 262.5;
	TX1(187, 1) = 267.1875;
	TX1(188, 1) = 271.875;
	TX1(189, 1) = 276.5625;
	TX1(190, 1) = 281.25;
	TX1(191, 1) = 285.9375;
	TX1(192, 1) = 290.625;
	TX1(193, 1) = 295.3125;
	TX1(194, 1) = 300;
	TX1(195, 1) = 0;
	TX1(196, 1) = 4.6875;
	TX1(197, 1) = 9.375;
	TX1(198, 1) = 14.0625;
	TX1(199, 1) = 18.75;
	TX1(200, 1) = 23.4375;
	TX1(201, 1) = 28.125;
	TX1(202, 1) = 32.8125;
	TX1(203, 1) = 37.5;
	TX1(204, 1) = 42.1875;
	TX1(205, 1) = 46.875;
	TX1(206, 1) = 51.5625;
	TX1(207, 1) = 56.25;
	TX1(208, 1) = 60.9375;
	TX1(209, 1) = 65.625;
	TX1(210, 1) = 70.3125;
	TX1(211, 1) = 75;
	TX1(212, 1) = 79.6875;
	TX1(213, 1) = 84.375;
	TX1(214, 1) = 89.0625;
	TX1(215, 1) = 93.75;
	TX1(216, 1) = 98.4375;
	TX1(217, 1) = 103.125;
	TX1(218, 1) = 107.8125;
	TX1(219, 1) = 112.5;
	TX1(220, 1) = 117.1875;
	TX1(221, 1) = 121.875;
	TX1(222, 1) = 126.5625;
	TX1(223, 1) = 131.25;
	TX1(224, 1) = 135.9375;
	TX1(225, 1) = 140.625;
	TX1(226, 1) = 145.3125;
	TX1(227, 1) = 150;
	TX1(228, 1) = 154.6875;
	TX1(229, 1) = 159.375;
	TX1(230, 1) = 164.0625;
	TX1(231, 1) = 168.75;
	TX1(232, 1) = 173.4375;
	TX1(233, 1) = 178.125;
	TX1(234, 1) = 182.8125;
	TX1(235, 1) = 187.5;
	TX1(236, 1) = 192.1875;
	TX1(237, 1) = 196.875;
	TX1(238, 1) = 201.5625;
	TX1(239, 1) = 206.25;
	TX1(240, 1) = 210.9375;
	TX1(241, 1) = 215.625;
	TX1(242, 1) = 220.3125;
	TX1(243, 1) = 225;
	TX1(244, 1) = 229.6875;
	TX1(245, 1) = 234.375;
	TX1(246, 1) = 239.0625;
	TX1(247, 1) = 243.75;
	TX1(248, 1) = 248.4375;
	TX1(249, 1) = 253.125;
	TX1(250, 1) = 257.8125;
	TX1(251, 1) = 262.5;
	TX1(252, 1) = 267.1875;
	TX1(253, 1) = 271.875;
	TX1(254, 1) = 276.5625;
	TX1(255, 1) = 281.25;
	TX1(256, 1) = 285.9375;
	TX1(257, 1) = 290.625;
	TX1(258, 1) = 295.3125;
	TX1(259, 1) = 300;
	TX1(260, 1) = 0;
	TX1(261, 1) = 4.6875;
	TX1(262, 1) = 9.375;
	TX1(263, 1) = 14.0625;
	TX1(264, 1) = 18.75;
	TX1(265, 1) = 23.4375;
	TX1(266, 1) = 28.125;
	TX1(267, 1) = 32.8125;
	TX1(268, 1) = 37.5;
	TX1(269, 1) = 42.1875;
	TX1(270, 1) = 46.875;
	TX1(271, 1) = 51.5625;
	TX1(272, 1) = 56.25;
	TX1(273, 1) = 60.9375;
	TX1(274, 1) = 65.625;
	TX1(275, 1) = 70.3125;
	TX1(276, 1) = 75;
	TX1(277, 1) = 79.6875;
	TX1(278, 1) = 84.375;
	TX1(279, 1) = 89.0625;
	TX1(280, 1) = 93.75;
	TX1(281, 1) = 98.4375;
	TX1(282, 1) = 103.125;
	TX1(283, 1) = 107.8125;
	TX1(284, 1) = 112.5;
	TX1(285, 1) = 117.1875;
	TX1(286, 1) = 121.875;
	TX1(287, 1) = 126.5625;
	TX1(288, 1) = 131.25;
	TX1(289, 1) = 135.9375;
	TX1(290, 1) = 140.625;
	TX1(291, 1) = 145.3125;
	TX1(292, 1) = 150;
	TX1(293, 1) = 154.6875;
	TX1(294, 1) = 159.375;
	TX1(295, 1) = 164.0625;
	TX1(296, 1) = 168.75;
	TX1(297, 1) = 173.4375;
	TX1(298, 1) = 178.125;
	TX1(299, 1) = 182.8125;
	TX1(300, 1) = 187.5;
	TX1(301, 1) = 192.1875;
	TX1(302, 1) = 196.875;
	TX1(303, 1) = 201.5625;
	TX1(304, 1) = 206.25;
	TX1(305, 1) = 210.9375;
	TX1(306, 1) = 215.625;
	TX1(307, 1) = 220.3125;
	TX1(308, 1) = 225;
	TX1(309, 1) = 229.6875;
	TX1(310, 1) = 234.375;
	TX1(311, 1) = 239.0625;
	TX1(312, 1) = 243.75;
	TX1(313, 1) = 248.4375;
	TX1(314, 1) = 253.125;
	TX1(315, 1) = 257.8125;
	TX1(316, 1) = 262.5;
	TX1(317, 1) = 267.1875;
	TX1(318, 1) = 271.875;
	TX1(319, 1) = 276.5625;
	TX1(320, 1) = 281.25;
	TX1(321, 1) = 285.9375;
	TX1(322, 1) = 290.625;
	TX1(323, 1) = 295.3125;
	TX1(324, 1) = 300;
	return TX1;
}

MatrixXd GetTX2()
{
	MatrixXd TX1(15, 2);

	TX1(0, 0) = 0;
	TX1(1, 0) = 0;
	TX1(2, 0) = 0;
	TX1(3, 0) = 0;
	TX1(4, 0) = 0;
	TX1(5, 0) = 0.432499999999999;
	TX1(6, 0) = 0.432499999999999;
	TX1(7, 0) = 0.432499999999999;
	TX1(8, 0) = 0.432499999999999;
	TX1(9, 0) = 0.432499999999999;
	TX1(10, 0) = 0.864999999999999;
	TX1(11, 0) = 0.864999999999999;
	TX1(12, 0) = 0.864999999999999;
	TX1(13, 0) = 0.864999999999999;
	TX1(14, 0) = 0.864999999999999;
	TX1(0, 1) = 0;
	TX1(1, 1) = 75;
	TX1(2, 1) = 150;
	TX1(3, 1) = 225;
	TX1(4, 1) = 300;
	TX1(5, 1) = 0;
	TX1(6, 1) = 75;
	TX1(7, 1) = 150;
	TX1(8, 1) = 225;
	TX1(9, 1) = 300;
	TX1(10, 1) = 0;
	TX1(11, 1) = 75;
	TX1(12, 1) = 150;
	TX1(13, 1) = 225;
	TX1(14, 1) = 300;
	return TX1;
}

int main()
{
	MatrixXd TX1 = GetTX2();
	MatrixXd CN = GetTX2();
	MatrixXd C(1, 2);
	MatrixXd A(1, 2);
	C << 1.73, 600;
	//A << 2, 64;
	A << 2, 4;
	MatrixXd D(TX1.rows(), TX1.rows());
	for (int i = 0; i < 1; i++)
	{
		printf("i=%i", i);
		ThrustLib::Gaussian::Gaussian2d(TX1, CN, A, C);
	}

	return 0;
}
